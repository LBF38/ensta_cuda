
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CUDA kernel to add elements of two arrays
__global__ void add(int n, float *x, float *y, float *z) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    z[index] = x[index] + y[index];
  }
}

// Function to initialize array with random values
void initialize(float *array, int n) {
  srand(time(0));
  for (int i = 0; i < n; i++) {
    array[i] = rand() / (float)RAND_MAX;
  }
}

int main(void) {
  int N = 1 << 20; // 1M elements

  // Allocate memory for arrays on the host
  float *x = (float *)malloc(N * sizeof(float));
  float *y = (float *)malloc(N * sizeof(float));
  float *z = (float *)malloc(N * sizeof(float)); // output array

  // Initialize arrays with random values
  initialize(x, N);
  initialize(y, N);

  // Allocate memory for arrays on the device
  float *d_x, *d_y, *d_z;
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));
  hipMalloc(&d_z, N * sizeof(float));

  // Copy input arrays from host to device
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Launch the CUDA kernel
  //   int threadsPerBlock = 256;
  //   int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  //   add<<<blocksPerGrid, threadsPerBlock>>>(N, d_x, d_y, d_z);
  add<<<1, 1>>>(N, d_x, d_y, d_z);

  // Copy the result array from device to host
  hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost);

  // Free memory on both host and device
  free(x);
  free(y);
  free(z);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  return 0;
}
