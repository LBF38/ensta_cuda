
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA kernel to multiply elements of two arrays
__global__ void multiply(int n, float *x, float *y, float *z) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    z[i] = x[i] * y[i];
}

// Function to initialize array with random numbers
void initializeArray(float *a, int n) {
  for (int i = 0; i < n; i++)
    a[i] = rand() / (float)RAND_MAX;
}

int main(int argc, char *argv[]) {
  if (argc != 4) {
    fprintf(stderr, "Usage: %s <input size> <grid size> <block size>\n",
            argv[0]);
    return 1;
  }

  int N = atoi(argv[1]);         // input size
  int gridSize = atoi(argv[2]);  // grid size
  int blockSize = atoi(argv[3]); // block size

  // Allocate memory for arrays on the host
  float *x = (float *)malloc(N * sizeof(float));
  float *y = (float *)malloc(N * sizeof(float));
  float *z = (float *)malloc(N * sizeof(float)); // output array

  // Initialize arrays with random values
  initializeArray(x, N);
  initializeArray(y, N);

  // Allocate memory for arrays on the device
  float *d_x, *d_y, *d_z;
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));
  hipMalloc(&d_z, N * sizeof(float));

  // Copy input arrays from host to device
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Launch the CUDA kernel
  multiply<<<gridSize, blockSize>>>(N, d_x, d_y, d_z);

  // Copy the result array from device to host
  hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost);

  // Free memory on both host and device
  free(x);
  free(y);
  free(z);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  return 0;
}
