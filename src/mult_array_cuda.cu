
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA kernel to multiply elements of two arrays
__global__ void multiply(int n, float *x, float *y, float *z) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    z[i] = x[i] * y[i];
}

// Function to initialize array with random numbers
void initializeArray(float *a, int n) {
  for (int i = 0; i < n; i++)
    a[i] = rand() / (float)RAND_MAX;
}

int main(void) {
  int N = 1 << 20; // 1M elements

  // Allocate memory for arrays on the host
  float *x = (float *)malloc(N * sizeof(float));
  float *y = (float *)malloc(N * sizeof(float));
  float *z = (float *)malloc(N * sizeof(float)); // output array

  // Initialize arrays with random values
  initializeArray(x, N);
  initializeArray(y, N);

  // Allocate memory for arrays on the device
  float *d_x, *d_y, *d_z;
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));
  hipMalloc(&d_z, N * sizeof(float));

  // Copy input arrays from host to device
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Launch the CUDA kernel
  //   int threadsPerBlock = 256;
  //   int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  //   multiply<<<blocksPerGrid, threadsPerBlock>>>(N, d_x, d_y, d_z);
  multiply<<<1, 1>>>(N, d_x, d_y, d_z);

  // Copy the result array from device to host
  hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost);

  // Free memory on both host and device
  free(x);
  free(y);
  free(z);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  return 0;
}
